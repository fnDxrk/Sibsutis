#include <hip/hip_runtime.h>
#include <iostream>

__global__ void hello_cuda()
{
    printf("Hello from CUDA kernel!\n");
}

int main()
{
    hello_cuda<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}

#include <hip/hip_runtime.h>
#include <iostream>

#define SIZE 8096

#define CUDA_CHECK_RETURN(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            const char* errStr; \
            hipDrvGetErrorString(err, &errStr); \
            std::cerr << "CUDA error: " << errStr << " at line " << __LINE__ << std::endl; \
            exit(1); \
        } \
    } while (0)


void fill_random(float *vec, size_t size) {
    for (size_t i = 0; i < size; i++) {
        vec[i] = rand() / RAND_MAX;
    }
}

void process_matrix_miltiplication() {
    srand(time(nullptr));
    CUDA_CHECK_RETURN(hipInit(0));

    hipDevice_t device;
    hipCtx_t context;
    hipModule_t module;
    hipFunction_t kernel;
    
    const char* ptx_file = 
        "/home/dxrk_/Documents/Sibsutis"
        "/Курс 3/Семестр 2/CUDA/Lab_9"
        "/cuda_driver_api/ptx/kernel.ptx";

    CUDA_CHECK_RETURN(hipDeviceGet(&device, 0));
    CUDA_CHECK_RETURN(hipCtxCreate(&context, 0, device));
    CUDA_CHECK_RETURN(hipModuleLoad(&module,ptx_file));
    CUDA_CHECK_RETURN(hipModuleGetFunction(
        &kernel, 
        module, 
        "matrix_multiplication"
    ));

    float *h_vec1, *h_vec2, *h_res;
    h_vec1 = new float[SIZE * SIZE];
    h_vec2 = new float[SIZE * SIZE];
    h_res = new float[SIZE * SIZE];

    fill_random(h_vec1, SIZE * SIZE);
    fill_random(h_vec2, SIZE * SIZE);
    
    hipDeviceptr_t d_vec1, d_vec2, d_res;
    CUDA_CHECK_RETURN(hipMalloc(
        &d_vec1, 
        SIZE * SIZE * sizeof(float)
    ));
    CUDA_CHECK_RETURN(hipMalloc(
        &d_vec2,
        SIZE * SIZE * sizeof(float)
    ));
    CUDA_CHECK_RETURN(hipMalloc(
        &d_res,
        SIZE * SIZE * sizeof(float)
    ));

    dim3 threads_per_block(16, 16);
    dim3 num_blocks(
        (SIZE + threads_per_block.x - 1) / threads_per_block.x,
        (SIZE + threads_per_block.y - 1) / threads_per_block.y
    );

    hipEvent_t start, end;
    CUDA_CHECK_RETURN(hipEventCreateWithFlags(&start, hipEventDefault));
    CUDA_CHECK_RETURN(hipEventCreateWithFlags(&end, hipEventDefault));


    size_t size = SIZE;
    void *args[] = {&d_vec1, &d_vec2, &d_res, &size};
    
    CUDA_CHECK_RETURN(hipEventRecord(start, 0));
    CUDA_CHECK_RETURN(hipModuleLaunchKernel(
        kernel,
        num_blocks.x, num_blocks.y, num_blocks.z,
        threads_per_block.x, threads_per_block.y, threads_per_block.z,
        0,            // sharedMemBytes (0 по умолчанию)
        nullptr,      // stream (nullptr для дефолтного)
        args,         // массив указателей на аргументы
        nullptr       // дополнительные параметры
    ));
    CUDA_CHECK_RETURN(hipEventRecord(end, 0));
    CUDA_CHECK_RETURN(hipEventSynchronize(end));
    CUDA_CHECK_RETURN(hipCtxSynchronize());

    CUDA_CHECK_RETURN(hipMemcpyDtoH(
        h_res, 
        d_res, 
        SIZE * SIZE * sizeof(float)
    ));

    float ellapsed_time;
    CUDA_CHECK_RETURN(hipEventElapsedTime(&ellapsed_time, start, end));
    printf("Время выполнения: %.4f мс\n", ellapsed_time);

    delete[] h_vec1;
    delete[] h_vec2;
    delete[] h_res;

    CUDA_CHECK_RETURN(hipFree(d_vec1));
    CUDA_CHECK_RETURN(hipFree(d_vec2));
    CUDA_CHECK_RETURN(hipFree(d_res));

    CUDA_CHECK_RETURN(hipEventDestroy(start));
    CUDA_CHECK_RETURN(hipEventDestroy(end));

    CUDA_CHECK_RETURN(hipModuleUnload(module));
    CUDA_CHECK_RETURN(hipCtxDestroy(context));

}

int main(int argc, char *argv[]) {
    process_matrix_miltiplication();
    return 0;
}

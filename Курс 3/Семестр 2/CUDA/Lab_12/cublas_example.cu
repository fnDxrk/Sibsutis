#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <iostream>
#include <chrono>

#define CUDA_CHECK_RETURN(value) { \
    hipError_t _m_cudaStat = value; \
    if (_m_cudaStat != hipSuccess) { \
        fprintf(stderr, "Ошибка %s в строке %d в файле %s\n", \
        hipGetErrorString(_m_cudaStat), __LINE__, __FILE__); \
        exit(1); \
    }}


#define M 4096
#define N 4096
#define K 4096

#define LDA M
#define LDB K
#define LDC M

void mm_no_tensor_cores(float *A,  float *B, float *C) {
    
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    const float alpha = 1.0;
    const float beta = 0.0;

    hipblasSgemm(
        cublas_handle, 
        HIPBLAS_OP_T, 
        HIPBLAS_OP_N,
        M, N, K,
        &alpha,
        A, LDA,
        B, LDB,
        &beta, 
        C, LDC 
    );

    hipblasDestroy(cublas_handle);
}

void mm_tensor_cores(float *A, float *B, float *C) {
    
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    const float alpha = 1.0;
    const float beta = 0.0;

    hipblasSetMathMode(cublas_handle, HIPBLAS_TENSOR_OP_MATH);

    hipblasSgemm(
        cublas_handle, 
        HIPBLAS_OP_T, 
        HIPBLAS_OP_N,
        M, N, K,
        &alpha,
        A, LDA,
        B, LDB,
        &beta, 
        C, LDC 
    );

    hipblasSetMathMode(cublas_handle, HIPBLAS_DEFAULT_MATH);

    hipblasDestroy(cublas_handle);
}

void fill_random(float *vec, int size) {
    for (int i = 0; i < size; i++)
        vec[i] = (float) rand() / RAND_MAX;
}

void process_mm() {

    float *h_A, *h_B, *h_C_no_tensors, *h_C_tensors;
    h_A = new float[M * K];
    h_B = new float[K * N];
    h_C_no_tensors = new float[M * N];
    h_C_tensors = new float[M * N];

    fill_random(h_A, M * K);
    fill_random(h_A, K * N);

    float *d_A, *d_B, *d_C;
    CUDA_CHECK_RETURN(hipMalloc(&d_A, M * K * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc(&d_B, K * N * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc(&d_C, M * N * sizeof(float)));

    CUDA_CHECK_RETURN(hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice));


    auto start = std::chrono::high_resolution_clock::now();
    mm_no_tensor_cores(d_A, d_B, d_C);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> no_tensors_time = end - start;
    CUDA_CHECK_RETURN(hipMemcpy(h_C_no_tensors, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));

    start = std::chrono::high_resolution_clock::now();
    mm_tensor_cores(d_A, d_B, d_C);
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> tensors_time = end - start;
    CUDA_CHECK_RETURN(hipMemcpy(h_C_tensors, d_C, M * N*sizeof(float), hipMemcpyDeviceToHost));

    printf("Время выполнения cuBLAS (без тензорных ядер): %.4f мс\n", no_tensors_time.count());
    printf("Время выполнения cuBLAS (с тензорными ядрами): %.4f мс\n", tensors_time.count());

}

int main(int argc, char *argv[]) {
    process_mm();
    return 0;
}
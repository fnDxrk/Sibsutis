#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void vectorAdd(const float* A, const float* B, float* C, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        if (idx == 3) {
            int* ptr = NULL;
            // Неправильное обращение к памяти: разыменование NULL.
            ptr[0] = 0;
        }
        C[idx] = A[idx] + B[idx];
    }
}

int main()
{
    const int N = 10;
    size_t size = N * sizeof(float);

    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    for (int i = 0; i < N; i++) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    float *d_A, *d_B, *d_C;
    hipError_t err;

    err = hipMalloc((void**)&d_A, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Ошибка выделения памяти на устройстве для A: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }
    err = hipMalloc((void**)&d_B, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Ошибка выделения памяти на устройстве для B: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }
    err = hipMalloc((void**)&d_C, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Ошибка выделения памяти на устройстве для C: %s\n", hipGetErrorString(err));
        return EXIT_FAILURE;
    }

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "Ошибка выполнения CUDA ядра: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("Результирующий вектор:\n");
    for (int i = 0; i < N; i++) {
        printf("%f ", h_C[i]);
    }
    printf("\n");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}

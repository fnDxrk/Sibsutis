#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <random>
#include <ctime>

#define CUDA_CHECK_RETURN(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            const char* errStr; \
            hipDrvGetErrorString(err, &errStr); \
            std::cerr << "CUDA error: " << errStr << " at line " << __LINE__ << std::endl; \
            exit(1); \
        } \
    } while (0)

void fill_random(float *vec, size_t size) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(0.0f, 1.0f);

    for (size_t i = 0; i < size; ++i) {
        vec[i] = dis(gen);
    }
}

void process_matrix_multiplication(size_t size) {
    CUDA_CHECK_RETURN(hipInit(0));

    hipDevice_t device;
    hipCtx_t context;
    hipModule_t module;
    hipFunction_t kernel;

    const char* ptx_file = "./kernel.ptx";

    CUDA_CHECK_RETURN(hipDeviceGet(&device, 0));
    CUDA_CHECK_RETURN(hipCtxCreate(&context, 0, device));
    CUDA_CHECK_RETURN(hipModuleLoad(&module, ptx_file));
    CUDA_CHECK_RETURN(hipModuleGetFunction(&kernel, module, "matrix_multiplication"));

    float *h_vec1, *h_vec2, *h_res;
    h_vec1 = new float[size * size];
    h_vec2 = new float[size * size];
    h_res = new float[size * size];

    fill_random(h_vec1, size * size);
    fill_random(h_vec2, size * size);

    hipDeviceptr_t d_vec1, d_vec2, d_res;
    CUDA_CHECK_RETURN(hipMalloc(&d_vec1, size * size * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc(&d_vec2, size * size * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc(&d_res, size * size * sizeof(float)));

    CUDA_CHECK_RETURN(hipMemcpyHtoD(d_vec1, h_vec1, size * size * sizeof(float)));
    CUDA_CHECK_RETURN(hipMemcpyHtoD(d_vec2, h_vec2, size * size * sizeof(float)));

    dim3 threads_per_block(16, 16);
    dim3 num_blocks((size + threads_per_block.x - 1) / threads_per_block.x,
                    (size + threads_per_block.y - 1) / threads_per_block.y);

    hipEvent_t start, end;
    CUDA_CHECK_RETURN(hipEventCreateWithFlags(&start, hipEventDefault));
    CUDA_CHECK_RETURN(hipEventCreateWithFlags(&end, hipEventDefault));

    size_t kernel_size = size;
    void *args[] = {&d_vec1, &d_vec2, &d_res, &kernel_size};

    const int iterations = 10;
    float total_time = 0.0f;
    for (int i = 0; i < iterations; ++i) {
        CUDA_CHECK_RETURN(hipEventRecord(start, 0));
        CUDA_CHECK_RETURN(hipModuleLaunchKernel(
            kernel,
            num_blocks.x, num_blocks.y, 1,
            threads_per_block.x, threads_per_block.y, 1,
            0, nullptr, args, nullptr));
        CUDA_CHECK_RETURN(hipEventRecord(end, 0));
        CUDA_CHECK_RETURN(hipEventSynchronize(end));
        float iter_time;
        CUDA_CHECK_RETURN(hipEventElapsedTime(&iter_time, start, end));
        total_time += iter_time;
    }
    CUDA_CHECK_RETURN(hipCtxSynchronize());

    CUDA_CHECK_RETURN(hipMemcpyDtoH(h_res, d_res, size * size * sizeof(float)));

    printf("Driver API, Matrix Size %lux%lu: %.4f мс (усреднено за %d итераций)\n",
           size, size, total_time / iterations, iterations);

    delete[] h_vec1;
    delete[] h_vec2;
    delete[] h_res;

    CUDA_CHECK_RETURN(hipFree(d_vec1));
    CUDA_CHECK_RETURN(hipFree(d_vec2));
    CUDA_CHECK_RETURN(hipFree(d_res));

    CUDA_CHECK_RETURN(hipEventDestroy(start));
    CUDA_CHECK_RETURN(hipEventDestroy(end));
    CUDA_CHECK_RETURN(hipModuleUnload(module));
    CUDA_CHECK_RETURN(hipCtxDestroy(context));
}

int main() {
    std::vector<size_t> sizes = {256, 512, 1024, 2048, 4096, 8192};
    for (size_t size : sizes) {
        process_matrix_multiplication(size);
    }
    return 0;
}